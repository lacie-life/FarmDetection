#include "hip/hip_runtime.h"
// This file is part of REMODE - REgularized MOnocular Depth Estimation.
//
// Copyright (C) 2014 Matia Pizzoli <matia dot pizzoli at gmail dot com>
// Robotics and Perception Group, University of Zurich, Switzerland
// http://rpg.ifi.uzh.ch
//
// REMODE is free software: you can redistribute it and/or modify it under the
// terms of the GNU General Public License as published by the Free Software
// Foundation, either version 3 of the License, or any later version.
//
// REMODE is distributed in the hope that it will be useful, but WITHOUT ANY
// WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
// FOR A PARTICULAR PURPOSE. See the GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.

#include <rmd/device_image.cuh>

namespace rmd
{

__global__
void copyKernel(
    const DeviceImage<float> *in_dev_ptr,
    DeviceImage<float> *out_dev_ptr)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if(x >= in_dev_ptr->width || y >= in_dev_ptr->height)
    return;

  const DeviceImage<float>  &img = *in_dev_ptr;
  DeviceImage<float> &copy = *out_dev_ptr;
  copy(x, y) = img(x, y);
}

void copy(
    const DeviceImage<float> &img,
    DeviceImage<float> &copy)
{
  // CUDA fields
  dim3 dim_block;
  dim3 dim_grid;
  dim_block.x = 16;
  dim_block.y = 16;
  dim_grid.x = (img.width  + dim_block.x - 1) / dim_block.x;
  dim_grid.y = (img.height + dim_block.y - 1) / dim_block.y;
  copyKernel<<<dim_grid, dim_block>>>(img.dev_ptr, copy.dev_ptr);
  hipDeviceSynchronize();
}

} // rmd namespace

